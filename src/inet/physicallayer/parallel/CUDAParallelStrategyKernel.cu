#include "hip/hip_runtime.h"
//
// Copyright (C) 2013 OpenSim Ltd.
//
// This program is free software; you can redistribute it and/or
// modify it under the terms of the GNU Lesser General Public License
// as published by the Free Software Foundation; either version 2
// of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with this program; if not, see <http://www.gnu.org/licenses/>.
//

#include "inet/physicallayer/parallel/CUDAParallelStrategyKernel.h"

namespace inet {

namespace physicallayer {

using namespace inet;

__global__ void deviceShift(void *buffer, int offset, int size)
{
    for (int i = 0; i < size - offset; i++)
        *((int8_t *)buffer + i) = *((int8_t *)buffer + offset + i);
}

__global__ void deviceComputeAllReceptionsForTransmission(double timeScale, int radioCount, double propagationSpeed, double pathLossAlpha,
                                                          double transmissionPower, double transmissionCarrierFrequency, cuda_simtime_t transmissionTime,
                                                          double transmissionPositionX, double transmissionPositionY, double transmissionPositionZ,
                                                          double *receptionPositionXs, double *receptionPositionYs, double *receptionPositionZs,
                                                          cuda_simtime_t *propagationTimes, cuda_simtime_t *receptionTimes, double *receptionPowers)
{
    int radioIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (radioIndex < radioCount)
    {
        double dx = transmissionPositionX - receptionPositionXs[radioIndex];
        double dy = transmissionPositionY - receptionPositionYs[radioIndex];
        double dz = transmissionPositionZ - receptionPositionZs[radioIndex];
        double distance = sqrt(dx * dx + dy * dy + dz * dz);

        cuda_simtime_t propagationTime = distance / propagationSpeed * timeScale;
        propagationTimes[radioIndex] = propagationTime;
        cuda_simtime_t receptionTime = transmissionTime + propagationTime;
        receptionTimes[radioIndex] = receptionTime;

        double waveLength = propagationSpeed / transmissionCarrierFrequency;
        // NOTE: this check allows to get the same result from the GPU and the CPU when the pathLossAlpha is exactly 2
        double ratio = waveLength / distance;
        double raisedRatio = pathLossAlpha == 2.0 ? ratio * ratio : pow(ratio, pathLossAlpha);
        double pathLoss = distance == 0.0 ? 1.0 : raisedRatio / (16.0 * M_PI * M_PI);
        double receptionPower = pathLoss * transmissionPower;
        receptionPowers[radioIndex] = receptionPower;
    }
}

__global__ void deviceComputeAllMinSNIRsForAllReceptions2(int transmissionCount, int radioCount, double backgroundNoisePower,
                                                          cuda_simtime_t *transmissionDurations, cuda_simtime_t *receptionTimes, double *receptionPowers,
                                                          double *minSNIRs)
{
    int receptionCount = transmissionCount * radioCount;
    int candidateTransmissionIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int candidateRadioIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int candidateReceptionIndex = candidateRadioIndex + radioCount * candidateTransmissionIndex;
    int newTransmissionIndex = transmissionCount - 1;
    cuda_simtime_t newTransmissionDuration = transmissionDurations[newTransmissionIndex];
    if (candidateTransmissionIndex < transmissionCount && candidateRadioIndex < radioCount)
    {
        cuda_simtime_t candidateTransmissionDuration = transmissionDurations[candidateTransmissionIndex];
        cuda_simtime_t candidateReceptionStartTime = receptionTimes[candidateReceptionIndex];
        cuda_simtime_t candidateReceptionEndTime = candidateReceptionStartTime + candidateTransmissionDuration;

        cuda_simtime_t newReceptionStartTime = receptionTimes[candidateRadioIndex + radioCount * newTransmissionIndex];
        cuda_simtime_t newReceptionEndTime = newReceptionStartTime + newTransmissionDuration;
        cuda_simtime_t interferenceStartTime = candidateReceptionStartTime > newReceptionStartTime ? candidateReceptionStartTime : newReceptionStartTime;
        cuda_simtime_t interferenceEndTime = candidateReceptionEndTime < newReceptionEndTime ? candidateReceptionEndTime : newReceptionEndTime;

        if (interferenceEndTime >= interferenceStartTime) {
            double maximumNoisePower = 0;
            for (int otherReceptionIndex = candidateRadioIndex; otherReceptionIndex < receptionCount; otherReceptionIndex += radioCount)
            {
                int otherTransmissionIndex = otherReceptionIndex / radioCount;
                cuda_simtime_t otherTransmissionDuration = transmissionDurations[otherTransmissionIndex];
                cuda_simtime_t otherReceptionStartTime = receptionTimes[otherReceptionIndex];
                cuda_simtime_t otherReceptionEndTime = otherReceptionStartTime + otherTransmissionDuration;
                bool isOtherStartOverlapping = interferenceStartTime <= otherReceptionStartTime && otherReceptionStartTime <= interferenceEndTime;
                bool isOtherEndOverlapping = interferenceStartTime <= otherReceptionEndTime && otherReceptionEndTime <= interferenceEndTime;
                if (isOtherStartOverlapping || isOtherEndOverlapping)
                {
                    double startNoisePower = 0;
                    double endNoisePower = 0;
                    for (int noiseReceptionIndex = candidateRadioIndex; noiseReceptionIndex < receptionCount; noiseReceptionIndex += radioCount)
                    {
                        if (noiseReceptionIndex != candidateReceptionIndex)
                        {
                            int noiseTransmissionIndex = noiseReceptionIndex / radioCount;
                            cuda_simtime_t noiseTransmissionDuration = transmissionDurations[noiseTransmissionIndex];
                            cuda_simtime_t noiseReceptionStartTime = receptionTimes[noiseReceptionIndex];
                            cuda_simtime_t noiseReceptionEndTime = noiseReceptionStartTime + noiseTransmissionDuration;
                            double noisePower = receptionPowers[noiseReceptionIndex];
                            if (isOtherStartOverlapping && noiseReceptionStartTime <= otherReceptionStartTime && otherReceptionStartTime <= noiseReceptionEndTime)
                                startNoisePower += noisePower;
                            if (isOtherEndOverlapping && noiseReceptionStartTime <= otherReceptionEndTime && otherReceptionEndTime <= noiseReceptionEndTime)
                                endNoisePower += noisePower;
                        }
                    }
                    if (isOtherStartOverlapping && startNoisePower > maximumNoisePower)
                        maximumNoisePower = startNoisePower;
                    if (isOtherEndOverlapping && endNoisePower > maximumNoisePower)
                        maximumNoisePower = endNoisePower;
                }
            }
            double candidateNoisePower = receptionPowers[candidateReceptionIndex];
            minSNIRs[candidateReceptionIndex] = candidateNoisePower / (maximumNoisePower + backgroundNoisePower);
        }
    }
}

__global__ void deviceComputeAllMinSNIRsForAllReceptions(int transmissionCount, int radioCount, double backgroundNoisePower,
                                                         cuda_simtime_t *transmissionDurations, cuda_simtime_t *receptionTimes, double *receptionPowers,
                                                         double *minSNIRs)
{
    int receptionCount = transmissionCount * radioCount;
    int candidateTransmissionIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int candidateRadioIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int candidateReceptionIndex = candidateRadioIndex + radioCount * candidateTransmissionIndex;
    if (candidateTransmissionIndex < transmissionCount && candidateRadioIndex < radioCount)
    {
        cuda_simtime_t candidateTransmissionDuration = transmissionDurations[candidateTransmissionIndex];
        cuda_simtime_t candidateReceptionStartTime = receptionTimes[candidateReceptionIndex];
        cuda_simtime_t candidateReceptionEndTime = candidateReceptionStartTime + candidateTransmissionDuration;
        double maximumNoisePower = 0;
        for (int otherReceptionIndex = candidateRadioIndex; otherReceptionIndex < receptionCount; otherReceptionIndex += radioCount)
        {
            int otherTransmissionIndex = otherReceptionIndex / radioCount;
            cuda_simtime_t otherTransmissionDuration = transmissionDurations[otherTransmissionIndex];
            cuda_simtime_t otherReceptionStartTime = receptionTimes[otherReceptionIndex];
            cuda_simtime_t otherReceptionEndTime = otherReceptionStartTime + otherTransmissionDuration;
            bool isOtherStartOverlapping = candidateReceptionStartTime <= otherReceptionStartTime && otherReceptionStartTime <= candidateReceptionEndTime;
            bool isOtherEndOverlapping = candidateReceptionStartTime <= otherReceptionEndTime && otherReceptionEndTime <= candidateReceptionEndTime;
            if (isOtherStartOverlapping || isOtherEndOverlapping)
            {
                double startNoisePower = 0;
                double endNoisePower = 0;
                for (int noiseReceptionIndex = candidateRadioIndex; noiseReceptionIndex < receptionCount; noiseReceptionIndex += radioCount)
                {
                    if (noiseReceptionIndex != candidateReceptionIndex)
                    {
                        int noiseTransmissionIndex = noiseReceptionIndex / radioCount;
                        cuda_simtime_t noiseTransmissionDuration = transmissionDurations[noiseTransmissionIndex];
                        cuda_simtime_t noiseReceptionStartTime = receptionTimes[noiseReceptionIndex];
                        cuda_simtime_t noiseReceptionEndTime = noiseReceptionStartTime + noiseTransmissionDuration;
                        double noisePower = receptionPowers[noiseReceptionIndex];
                        if (isOtherStartOverlapping && noiseReceptionStartTime <= otherReceptionStartTime && otherReceptionStartTime <= noiseReceptionEndTime)
                            startNoisePower += noisePower;
                        if (isOtherEndOverlapping && noiseReceptionStartTime <= otherReceptionEndTime && otherReceptionEndTime <= noiseReceptionEndTime)
                            endNoisePower += noisePower;
                    }
                }
                if (isOtherStartOverlapping && startNoisePower > maximumNoisePower)
                    maximumNoisePower = startNoisePower;
                if (isOtherEndOverlapping && endNoisePower > maximumNoisePower)
                    maximumNoisePower = endNoisePower;
            }
        }
        double candidateNoisePower = receptionPowers[candidateReceptionIndex];
        minSNIRs[candidateReceptionIndex] = candidateNoisePower / (maximumNoisePower + backgroundNoisePower);
    }
}

void hostShift(void *buffer, int offset, int size)
{
    dim3 blockSize;
    blockSize.x = 1;
    dim3 gridSize;
    gridSize.x = 1;
    deviceShift<<<gridSize, blockSize>>>(buffer, offset, size);
}

void hostComputeAllReceptionsForTransmission(double timeScale, int radioCount, double propagationSpeed, double pathLossAlpha,
                                             double transmissionPower, double transmissionCarrierFrequency, cuda_simtime_t transmissionTime,
                                             double transmissionPositionX, double transmissionPositionY, double transmissionPositionZ,
                                             double *receptionPositionXs, double *receptionPositionYs, double *receptionPositionZs,
                                             cuda_simtime_t *propagationTimes, cuda_simtime_t *receptionTimes, double *receptionPowers)
{
    dim3 blockSize;
    blockSize.x = 4;
    dim3 gridSize;
    gridSize.x = radioCount / blockSize.x + 1;
    deviceComputeAllReceptionsForTransmission<<<gridSize, blockSize>>>(
            timeScale, radioCount, propagationSpeed, pathLossAlpha,
            transmissionPower, transmissionCarrierFrequency, transmissionTime,
            transmissionPositionX, transmissionPositionY, transmissionPositionZ,
            receptionPositionXs, receptionPositionYs, receptionPositionZs,
            propagationTimes, receptionTimes, receptionPowers);
}

void hostComputeAllMinSNIRsForAllReceptions2(int transmissionCount, int radioCount, double backgroundNoisePower,
                                             cuda_simtime_t *transmissionDurations, cuda_simtime_t *receptionTimes, double *receptionPowers,
                                             double *minSNIRs)
{
    dim3 blockSize;
    blockSize.x = 4;
    blockSize.y = 4;
    dim3 gridSize;
    gridSize.x = transmissionCount / blockSize.x + 1;
    gridSize.y = radioCount / blockSize.y + 1;
    deviceComputeAllMinSNIRsForAllReceptions2<<<gridSize, blockSize>>>(
        transmissionCount, radioCount, backgroundNoisePower,
        transmissionDurations, receptionTimes, receptionPowers,
        minSNIRs);
}

void hostComputeAllMinSNIRsForAllReceptions(int transmissionCount, int radioCount, double backgroundNoisePower,
                                            cuda_simtime_t *transmissionDurations, cuda_simtime_t *receptionTimes, double *receptionPowers,
                                            double *minSNIRs)
{
    dim3 blockSize;
    blockSize.x = 4;
    blockSize.y = 4;
    dim3 gridSize;
    gridSize.x = transmissionCount / blockSize.x + 1;
    gridSize.y = radioCount / blockSize.y + 1;
    deviceComputeAllMinSNIRsForAllReceptions<<<gridSize, blockSize>>>(
        transmissionCount, radioCount, backgroundNoisePower,
        transmissionDurations, receptionTimes, receptionPowers,
        minSNIRs);
}

} // namespace physicallayer

} // namespace inet

